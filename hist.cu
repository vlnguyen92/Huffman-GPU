#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


//#include "../cuda_by_example/common/book.h"
#include <iostream>
#include <stdio.h>

#define CHECK(ans) {gpuAssert((ans),__FILE__,__LINE__);}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if(code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n",hipGetErrorString(code),file, line);
        if(abort) exit(code);
    }
}

using namespace std;

#define SIZE    (100*1024*1024)


__global__ void histo_kernel( unsigned char *buffer,
        long size,
        unsigned int *histo ) {

    __shared__  unsigned int temp[256];

    temp[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
    while (i < size) {
        atomicAdd( &temp[buffer[i]], 1 );
        i += offset;
    }

    __syncthreads();
    atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}

int run(char* file, unsigned int* freq, unsigned int memSize, unsigned int *source) {

//    FILE *f = fopen(file,"rb");
//    if (!f) {perror(file); exit(1);}
//    fseek(f,0,SEEK_END);
//    unsigned int memSize = ftell(f);
//    fclose(f);

//    unsigned int* source;// = (unsigned int*) malloc (memSize);
//    hipHostAlloc((void**)&source,memSize,hipHostMallocDefault);

    FILE *f = fopen(file,"rb");
    if (!f) {perror(file); exit(1);}
    fseek(f,0,SEEK_SET);
    fread(source,1,memSize,f);

    fclose(f);

    unsigned char *buffer = (unsigned char*)source;

    hipDeviceProp_t  prop;
    ( hipGetDeviceProperties( &prop, 0 ) );
    int blocks = prop.multiProcessorCount;
    if(!prop.deviceOverlap)
    {
        cout << "No overlaps, so no speedup from streams" << endl;
        return 0;
    }

    // allocate memory on the GPU for the file's data
    int partSize = memSize/32;
    int totalNum = memSize/sizeof(unsigned int);
    int partialNum = partSize/sizeof(unsigned int);

    unsigned char *dev_buffer0; 
    unsigned char *dev_buffer1;
    unsigned int *dev_histo;
    hipMalloc( (void**)&dev_buffer0, partSize ) ;
    hipMalloc( (void**)&dev_buffer1, partSize ) ;
    hipMalloc( (void**)&dev_histo,
            256 * sizeof( int ) ) ;
    hipMemset( dev_histo, 0,
            256 * sizeof( int ) ) ;
    hipStream_t stream0, stream1;
    CHECK(hipStreamCreate(&stream0));
    CHECK(hipStreamCreate(&stream1));
    hipEvent_t     start, stop;
    ( hipEventCreate( &start ) );
    ( hipEventCreate( &stop ) );
    ( hipEventRecord( start, 0 ) );


    for(int i = 0; i < totalNum; i+=partialNum*2)
    {

        CHECK(hipMemcpyAsync(dev_buffer0, buffer+i, partSize, hipMemcpyHostToDevice,stream0));
        CHECK(hipMemcpyAsync(dev_buffer1, buffer+i+partialNum, partSize, hipMemcpyHostToDevice,stream1));


        // kernel launch - 2x the number of mps gave best timing
        histo_kernel<<<blocks*2,256,0,stream0>>>( dev_buffer0, partSize, dev_histo );
        histo_kernel<<<blocks*2,256,0,stream1>>>( dev_buffer1, partSize, dev_histo );
    }
    CHECK(hipStreamSynchronize(stream0));
    CHECK(hipStreamSynchronize(stream1));
    hipMemcpy( freq, dev_histo, 256 * sizeof( int ), hipMemcpyDeviceToHost );
    ( hipEventRecord( stop, 0 ) );
    ( hipEventSynchronize( stop ) );
    float   elapsedTime;
    ( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );



    // get stop time, and display the timing results
    //    printf( "Time to generate:  %3.1f ms\n", elapsedTime );

    //    long histoCount = 0;
    //    for (int i=0; i<256; i++) {
    //        histoCount += histo[i];
    //    }
    //    printf( "Histogram Sum:  %ld\n", histoCount );
    //
    // verify that we have the same counts via CPU
    //    for (int i=0; i<memSize; i++)
    //        freq[buffer[i]]--;
    //    for (int i=0; i<256; i++) {
    //        if (freq[i] != 0)
    //            printf( "Failure at %d!\n", i );
    //    }

    //    for(int i = 0; i < 256; i++)
    //      cout << freq[i] << endl;
    //    cout << "HERE" << endl;

    hipFree( dev_histo );
    hipFree( dev_buffer0 );
    hipFree( dev_buffer1 );
//    hipHostFree(buffer);
    return 0;
}
