/*
 * PAVLE - Parallel Variable-Length Encoder for CUDA. Main file.
 *
 * Copyright (C) 2009 Ana Balevic <ana.balevic@gmail.com>
 * All rights reserved.
 *
 * This program is free software; you can redistribute it and/or modify it under the terms of the
 * MIT License. Read the full licence: http://www.opensource.org/licenses/mit-license.php
 *
 * If you find this program useful, please contact me and reference PAVLE home page in your work.
 * 
 */

#include "stdafx.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "cuda_helpers.h"
#include "uint256.h"
#include "print_helpers.h"
#include "comparison_helpers.h"
#include "stats_logger.h"
#include "load_data.h"
//#include "cutil.h"

#include "vlc_kernel_sm64huff.cu"
#include "vlc_kernel_sm32.cu"
#include "scan.cu"
#include "pack_kernels.cu"
#include "cpuencode.cpp"

void runVLCTest(char *file_name, uint num_block_threads, uint num_blocks=1);

extern "C" void cpu_vlc_encode(unsigned int* indata, unsigned int num_elements, unsigned int* outdata, unsigned int *outsize, uint256 *codewords, unsigned int* codewordlens);
extern "C" void cpu_vlc_encode_lame(unsigned int* sourceData, unsigned int num_elements, 
					unsigned int* destData, unsigned int *outsize, 
					unsigned int *codewords, unsigned int* codewordlens); 

int main(int argc, char* argv[]){
	if(!InitCUDA()) { return 0;	}
	unsigned int num_block_threads = 256;
	if (argc > 1)
		for (int i=1; i<argc; i++)
			runVLCTest(argv[i], num_block_threads);
	else {runVLCTest(NULL, num_block_threads, 1024);	}
	CUDA_SAFE_CALL(hipDeviceReset());
	return 0;
}

//void runVLCTest(unsigned int num_blocks, unsigned int num_block_threads, unsigned int nnsymbols) { 
void runVLCTest(char *file_name, uint num_block_threads, uint num_blocks) {
	printf("CUDA! Starting VLC Tests!\n");
	unsigned int num_elements; //uint num_elements = num_blocks * num_block_threads; 
	unsigned int mem_size; //uint mem_size = num_elements * sizeof(int); 
	unsigned int symbol_type_size = sizeof(int);
	//////// LOAD DATA ///////////////
	double H; // entropy
	initParams(file_name, num_block_threads, num_blocks, num_elements, mem_size, symbol_type_size);
	printf("Parameters: num_elements: %d, num_blocks: %d, num_block_threads: %d\n----------------------------\n", num_elements, num_blocks, num_block_threads);
  printf("input size: %d\n", mem_size);
	////////LOAD DATA ///////////////
	uint	*sourceData =	(uint*) malloc(mem_size);
	uint	*destData   =	(uint*) malloc(mem_size);
	uint	*crefData   =	(uint*) malloc(mem_size*16);

	uint256	*codewords	   = (uint256*) malloc(NUM_SYMBOLS*sizeof(uint256));
	uint	*codewordlens  = (uint*) malloc(NUM_SYMBOLS*symbol_type_size);

	uint	*cw32 =		(uint*) malloc(mem_size);
	uint	*cw32len =	(uint*) malloc(mem_size);
	uint	*cw32idx =	(uint*) malloc(mem_size);

	uint	*cindex2=	(uint*) malloc(num_blocks*sizeof(int));

	memset(sourceData,   0, mem_size);
	memset(destData,     0, mem_size);
	memset(crefData,     0, mem_size);
	memset(cw32,         0, mem_size);
	memset(cw32len,      0, mem_size);
	memset(cw32idx,      0, mem_size);
	memset(codewords,    0, NUM_SYMBOLS*symbol_type_size);
	memset(codewordlens, 0, NUM_SYMBOLS*symbol_type_size);
	memset(cindex2, 0, num_blocks*sizeof(int));
	//////// LOAD DATA ///////////////
	loadData(file_name, sourceData, codewords, codewordlens, num_elements, mem_size, H);
	//printf("Codewords 32bit:\n");
	//print_array_in_hex(codewords, 256);
	//print_array<uint>(codewordlens, 256);
	//////// LOAD DATA ///////////////

	unsigned int	*d_sourceData, *d_destData, *d_destDataPacked;
	unsigned int	*d_codewordlens;
  uint256       *d_codewords;
	unsigned int	*d_cw32, *d_cw32len, *d_cw32idx, *d_cindex, *d_cindex2;

	CUDA_SAFE_CALL(hipMalloc((void**) &d_sourceData,		  mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_destData,			  mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_destDataPacked,	  mem_size));

	CUDA_SAFE_CALL(hipMalloc((void**) &d_codewords,		  NUM_SYMBOLS*sizeof(uint256)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_codewordlens,		  NUM_SYMBOLS*symbol_type_size));

	CUDA_SAFE_CALL(hipMalloc((void**) &d_cw32,				  mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_cw32len,			  mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_cw32idx,			  mem_size));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_cindex,         num_blocks*sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_cindex2,        num_blocks*sizeof(unsigned int)));

	CUDA_SAFE_CALL(hipMemcpy(d_sourceData,		sourceData,		mem_size,		hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_codewords,		codewords,		NUM_SYMBOLS*symbol_type_size,	hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_codewordlens,	codewordlens,	NUM_SYMBOLS*symbol_type_size,	hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_destData,		destData,		mem_size,		hipMemcpyHostToDevice));

    dim3 grid_size(num_blocks,1,1);
    dim3 block_size(num_block_threads, 1, 1);
	// unsigned int sm_block_size_for_cwlens = num_block_threads * sizeof(unsigned int);
	unsigned int sm_size; 
//  printf("size: %d\n", sizeof(uint256));
	unsigned int NT = 10; //number of runs for each execution time
	float ktime = 0.0f;
	unsigned int timer = 0;
//	CUT_SAFE_CALL(cutCreateTimer(&timer));


	//////////////////* CPU ENCODER *///////////////////////////////////
	ktime = 0;
//	CUT_SAFE_CALL(cutResetTimer(timer));
//	CUT_SAFE_CALL(cutStartTimer(timer));
	unsigned int refbytesize;
	cpu_vlc_encode((unsigned int*)sourceData, num_elements, (unsigned int*)crefData,  &refbytesize, codewords, codewordlens);
//  print_array_ints_as_bits(codewords,NUM_SYMBOLS);
//  print_array_ints_as_bits(codewordlens,NUM_SYMBOLS);
	//cpu_vlc_encode_lame((unsigned int*)sourceData, num_elements, (unsigned int*)crefData,  &refbytesize, codewords, codewordlens);
//	CUT_SAFE_CALL(cutStopTimer(timer));
//	ktime = cutGetTimerValue(timer);
	printdbg_data_bin("input.txt", sourceData, num_elements); 
	printf("CPU Encoding time (CPU): %f (ms)\n", ktime);
	printf("CPU Encoded to %d [B]\n", refbytesize);
	unsigned int num_ints = refbytesize/4 + ((refbytesize%4 ==0)?0:1);
	printdbg_data_bin("cpuout.txt", crefData, num_ints);
	//////////////////* END CPU *///////////////////////////////////


#if 0
	//////////////////* SM32 KERNEL *///////////////////////////////////
    grid_size.x		= num_blocks;
    block_size.x	= num_block_threads;
	sm_size			= block_size.x*sizeof(unsigned int);
#ifdef CACHECWLUT
	sm_size			= 2*NUM_SYMBOLS*sizeof(int) + block_size.x*sizeof(unsigned int);
#endif
	ktime			= 0;
//	CUT_SAFE_CALL(cutResetTimer(timer));
//	CUT_SAFE_CALL(cutStartTimer(timer));
	for (int i=0; i<NT; i++) {
	vlc_encode_kernel_sm32<<<grid_size, block_size, sm_size>>>(d_sourceData, d_codewords, d_codewordlens,  
																					d_cw32, d_cw32len, d_cw32idx, 
																d_destData, d_cindex); //testedOK2
	}
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed\n");
//	CUT_SAFE_CALL(cutStopTimer(timer));
//	ktime += cutGetTimerValue(timer);
	printf("GPU Encoding time (SM32): %f (ms)\n", ktime/NT);
	//////////////////* END KERNEL *///////////////////////////////////
#endif



#if 1
	//////////////////* SM64HUFF KERNEL *///////////////////////////////////
    grid_size.x		= num_blocks;
    block_size.x	= num_block_threads;
	sm_size			= block_size.x*sizeof(unsigned int);
#ifdef CACHECWLUT
	sm_size			= 2*NUM_SYMBOLS*sizeof(int) + block_size.x*sizeof(unsigned int);
#endif
	ktime			= 0;
//	CUT_SAFE_CALL(cutResetTimer(timer));
//	CUT_SAFE_CALL(cutStartTimer(timer));
	for (int i=0; i<NT; i++) {
	vlc_encode_kernel_sm64huff<<<grid_size, block_size, sm_size>>>(d_sourceData, d_codewords, d_codewordlens,  
																					d_cw32, d_cw32len, d_cw32idx, 
																d_destData, d_cindex); //testedOK2
	}
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed\n");
//	CUT_SAFE_CALL(cutStopTimer(timer));
//	ktime += cutGetTimerValue(timer);
//	printf("GPU Encoding time (SM64HUFF): %f (ms)\n", ktime/NT);
	//////////////////* END KERNEL *///////////////////////////////////
#endif




#if 1
	unsigned int num_scan_elements = grid_size.x;
	preallocBlockSums(num_scan_elements);
	hipMemset(d_destDataPacked, 0, mem_size);
	printf("Num_blocks to be passed to scan is %d.\n", num_scan_elements);
	prescanArray(d_cindex2, d_cindex, num_scan_elements);

	pack2<<< num_scan_elements/16, 16>>>((unsigned int*)d_destData, d_cindex, d_cindex2, (unsigned int*)d_destDataPacked, num_elements/num_scan_elements);
	CUT_CHECK_ERROR("Pack2 Kernel execution failed\n");
	deallocBlockSums();

	CUDA_SAFE_CALL(hipMemcpy(destData, d_destDataPacked, mem_size, hipMemcpyDeviceToHost));
	compare_vectors((unsigned int*)crefData, (unsigned int*)destData, num_ints);
	printdbg_data_bin("cpuout.txt", crefData, num_ints); 
	printdbg_data_bin("gpuout_dpt2.txt", destData, num_ints); 
	//CUDA_SAFE_CALL(hipMemcpy(cindex2, d_cindex2, num_blocks*sizeof(int), hipMemcpyDeviceToHost));
	//printdbg_data_int("blockscan.txt", cindex2, num_blocks); 
	//CUDA_SAFE_CALL(hipMemcpy(cw32, d_cw32, mem_size, hipMemcpyDeviceToHost));
	//CUDA_SAFE_CALL(hipMemcpy(cw32len, d_cw32len, mem_size, hipMemcpyDeviceToHost));
	//CUDA_SAFE_CALL(hipMemcpy(cw32idx, d_cw32idx, mem_size, hipMemcpyDeviceToHost));
	//printdbg_gpu_data_detailed2("gpuout_dpt2detailed.txt", cw32, cw32len, cw32idx, num_ints);
#endif

	free(sourceData); free(destData);  	free(codewords);  	free(codewordlens); free(cw32);  free(cw32len); free(crefData); 
	CUDA_SAFE_CALL(hipFree(d_sourceData)); 	CUDA_SAFE_CALL(hipFree(d_destData)); CUDA_SAFE_CALL(hipFree(d_destDataPacked));
	CUDA_SAFE_CALL(hipFree(d_codewords)); 		CUDA_SAFE_CALL(hipFree(d_codewordlens));
	CUDA_SAFE_CALL(hipFree(d_cw32)); 		CUDA_SAFE_CALL(hipFree(d_cw32len)); 	CUDA_SAFE_CALL(hipFree(d_cw32idx)); 
	CUDA_SAFE_CALL(hipFree(d_cindex)); CUDA_SAFE_CALL(hipFree(d_cindex2));
	free(cindex2);	
}

